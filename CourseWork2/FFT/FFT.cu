#include <hip/hip_runtime.h> 
 
#include <builtin_types.h> 
#include "float.h"

#define _SIZE_T_DEFINED 
#ifndef __CUDACC__ 
#define __CUDACC__ 
#endif 
#ifndef __cplusplus 
#define __cplusplus 
#endif

extern "C" {
	// Device code
	__global__ void FFT(short* A, int N)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i < N)
			A[i] = A[i] + A[i];
	}
}